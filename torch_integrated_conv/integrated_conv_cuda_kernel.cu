#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>  // for getCurrentCUDAStream()
#include <hip/hip_cooperative_groups.h>




/*
  Tiled summing reduction within a warp.  Requires that the thread-block
  be 1-dimensional, i.e.  blockDim.y == blockDim.z == 1.  Does not use
  __syncthreads, so it is safe to call in a subset of threads.
  TODO: we can in principle do this without a buffer, using __shfl_down()
  (see here https://sodocumentation.net/cuda/topic/6566/parallel-reduction--e-g--how-to-sum-an-array-)
  if CC >= 3.0.

  Args:
      threads_per_tile:  Must be a power of 2 in the interval [1,32].  Summation is
                         within blocks of threads of this size.
       buf:              Pointer to the start of a __shared__ buffer of size
                         blockDim.x, to be used as a temporary within this function.
       val:              The value to be summed
   Return:
       Threads where blockDim.x % threads_per_tile == 0 will return the sum:
         \sum_{i=0}^{threads_per_tile-1} [val in thread threadIdx.x + i]
       Return value in other threads is undefined.
 */
template <typename scalar_t>
__forceinline__ __device__ scalar_t tiled_warp_reduce_sum(int threads_per_tile,
                                                          __volatile__ scalar_t *buf,
                                                          scalar_t val) {
  // Each iteration halves the number of active threads
  // Each thread adds its partial sum[i] to sum[lane+i]
  for (int i = threads_per_tile / 2; i > 0; i /= 2) {
    buf[threadIdx.x] = val;
    if (threadIdx.x % threads_per_tile < i)
      val += buf[threadIdx.x + i];
  }
  return val; // Only threads with threadIdx.x % threads_per_tile == 0 will
              // return the full sums of their tiles.
}

/*
  Forward of integrated_conv.  Each thread group handles a single channel (equal
  to blockIdx.x), and loops over patches of the output and over the image n
  within the batch (different thread groups may be responsible for different
  subsets of patches and/or images, see docs of gridDim below).

  Template args:
      scalar_t: the floating-point type, e.g. float, double, maybe half.

  Args:
      input:  input image, shape (N, 2*C, H, W)
      pos_add:  positional encoding, additive part,  shape (C, kH, kW)
      pos_mul:  positional encoding, multiplicative part, shape (C, kH, kW)
      output:   output image, shape (N, 2*C, H, W)
   Note: kH and kW must both be odd so that it's clear how to zero-pad.

  The thread-block should have one dimension (x); blockDim.x should equal
  some small power of 2 (threads_per_opixel) times the output-patch size which is
  opatchH * opatchW (the output-patch height and width).  We expect
  threads_per_opixel to be 1, 2, or 4; we use a linear summation to sum up the
  different threads' partial sums, and if threads_per_opixel gets larger we'd
  need to make this a logarithmic reduction.

   The requirements on the grid dimension are:
       gridDim.x == num-channels C (required)
       gridDim.y <= num-patches per image (recommended)
       gridDim.z <= batch-size N (recommended)
  When we invoke this kernel, we'll invoke it as:
   integrated_conv_forward<<<gridDim, blockDim, bytesShared, stream>>>
  where bytesShared is the number of bytes needed in `extern_buf`:
    bytesShared = sizeof(shared_t) * numel, where
    numel = 2 * (kH * kW) + max(blockDim.x, (opatchH + kH - 1) * (patchW + kW - 1))
 */
extern __shared__ int extern_buf[];

template <typename scalar_t>
__global__
void integrated_conv_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4> input,  // N, 2*C, H, W
    torch::PackedTensorAccessor32<scalar_t, 3> pos_add,  // C, kH, kW
    torch::PackedTensorAccessor32<scalar_t, 3> pos_mul,  // C, kH, kW
    torch::PackedTensorAccessor32<scalar_t, 4> output,  // N, C, H, W
    int opatchH,  // output-patch height,
    int opatchW  // output-patch width
                             ) {
  const int H = input.size(2),
      W = input.size(3),
      kH = pos_add.size(1),
      kW = pos_add.size(2),
      npatchH = (H + opatchH - 1) / opatchH,  // num patches in vertical dim
      npatchW = (W + opatchW - 1) / opatchW,  // num patches in horizontal dim
      npatch = npatchH * npatchW;  // total number of patches per image

  // Channel index.
  const int c = blockIdx.x;
  // We don't need to check the range of `c` because we set gridDim.x to the
  // exact number of channels.

  const int ipatchH = opatchH + kH - 1,
      ipatchW = opatchW + kW - 1,
      ipatch_size = ipatchH * ipatchW,
      opatch_size = opatchH * opatchW;

  // `extern_buf` is general-purpose shared memory, which we'll divide between
  // pos_add, pos_mul and src_img_buf to be shared between the src image size
  // (ipatch_size) and the number of threads (blockDim.x)

  // these are pointers to __shared__ memory; the compiler should
  // be able to figure this out.
  scalar_t
      *pos_add_buf = (scalar_t*)extern_buf,     // pos_add positional-encoding / kernel parameters,
                                // indexed [kh*kW + kw] where kh and kw are vertical
                                // and horizontal positions in the kernel.
      *pos_mul_buf = pos_add_buf + (kH * kW), // pos_mul positional-encoding / kernel parameters,
                                              // indexed [kh*kW + kw] where kh and kw are vertical
                                              // and horizontal positions in the kernel.
      *src_img_buf = pos_mul_buf + (kH * kW);    // version of input image that relates to source position,
                             // of size [ipatch_size], indexed [h*ipatchW + w]...
                             // note, the 'h' and 'w' indexes are into the zero-padded input
                             // image.


  int threads_per_opixel = blockDim.x / opatch_size;
  assert(blockDim.x == opatch_size * threads_per_opixel);

  // pos_in_patch will be interpreted as h_in_patch * opatchW + w_in_patch.
  int pos_in_patch = threadIdx.x / threads_per_opixel;

  // Load parts of the kernel parameters pos_add and pos_mul into shared memory,
  // in pos_add_buf and pos_mul_buf
  for (int i = threadIdx.x; i < kH * kW; i += blockDim.x) {
    int kh = i / kW,
        kw = i % kW;
    pos_add_buf[i] = pos_add[c][kh][kw];
    pos_mul_buf[i] = pos_mul[c][kh][kw];
  }

  // n is the index within the batch.  Loop to make sure we cover all images in
  // the batch.  input.size(0) is the batch size N.  All threads in the thread-block
  // loop the same number of times.
  for (int n = blockIdx.z; n < input.size(0); n += gridDim.z) {

    // Loop over the patch within the output image.  All threads in the
    // thread-block loop the same number of times.
    for (int patch_idx = blockIdx.y; patch_idx < npatch; patch_idx += gridDim.y) {
      // (patch_h_offset, patch_w_offset) are the (vertical, horizontal) indexes
      // of the lowest-numbered pixel in the patch of output that this thread
      // block is responsible for.
      int patch_h_offset = (patch_idx / npatchW) * opatchH,
          patch_w_offset = (patch_idx % npatchW) * opatchW;

      // This __syncthreads() is only necessary if we have already looped at
      // least once over n or patch_idx: it's in case other threads are still
      // using the `src_img_buf` buffer for something else.
      __syncthreads();

      // Load the 'src' part of the input patch; the size of this is the size of
      // the output patch plus a border of sizes kH//2, kW//2 on each side.
      for (int i = threadIdx.x; i < ipatch_size; i += blockDim.x) {
        int h_in_kernel = i / ipatchW,
            w_in_kernel = i % ipatchW;
        int src_h = patch_h_offset + h_in_kernel - (kH / 2),  // kH / 2 is offset due to padding
            src_w = patch_w_offset + w_in_kernel - (kW / 2);
        scalar_t src_val = scalar_t(0);
        if ((unsigned int)src_h < (unsigned int)H &&  // h >= 0 && h < H
            (unsigned int)src_w < (unsigned int)W)    // w >= 0 && w < W
          src_val = input[n][c][src_h][src_w];
        src_img_buf[i] = src_val;
      }
      // make sure all threads have written to `src_img_buf`
      __syncthreads();


      // 'h' and 'w' are the positions within the output image, that this tile
      // of size threads_per_opixel is responsible for.
      int h = patch_h_offset + pos_in_patch / opatchW,
          w = patch_w_offset + pos_in_patch % opatchW;

      // The "destination" pixel; this is an input.  It gets added to each
      // src pixel, prior to the relu, in the loop below.
      scalar_t dest_val = scalar_t(0);
      if (h < H && w < W) {
        // Several threads (within the same tile, which implies the same warp)
        // may load the same value here, but I believe the device's memory
        // subsystem handles this well enough that we can just ignore the issue
        // rather than try to optimize it.
        // https://forums.developer.nvidia.com/t/accessing-same-global-memory-address-within-warps/66574
        int C = input.size(1) / 2;
        dest_val = input[n][c + C][h][w];  // else 0.
      }

      // `sum` is the partial sum that this thread computes; we'll sum this over
      // the `threads_per_opixel` threads in the tile to get the output pixel
      // value.
      scalar_t sum = 0.0;

      for (int pos_in_kernel = threadIdx.x % threads_per_opixel;
           pos_in_kernel < (kH * kW);
           pos_in_kernel += threads_per_opixel) {
        int h_in_kernel = pos_in_kernel / kW,
            w_in_kernel = pos_in_kernel % kW;
        // Note: this is actually more like cross-correlation, as we don't
        // have a negative sign on the h and w indexes in the kernel.
        // Also note: we already took care of padding and the associated
        // offsets of -(kH / 2) and -(kW / 2).
        int h_in_src_patch = (pos_in_patch / opatchW) + h_in_kernel,
            w_in_src_patch = (pos_in_patch % opatchW) + w_in_kernel;
        scalar_t src_val = src_img_buf[h_in_src_patch * ipatchW + w_in_src_patch],
            pos_add_val = pos_add_buf[pos_in_kernel];
        scalar_t relu = (src_val + dest_val + pos_add_val);
        if (relu > 0.0)
          sum += relu * pos_mul_buf[pos_in_kernel];
      }
      // Aggregate `sum` over threads
      sum = tiled_warp_reduce_sum(threads_per_opixel, src_img_buf, sum);
      if (threadIdx.x % threads_per_opixel == 0 && h < H && w < W) {
        output[n][c][h][w] = sum;
      }
    }
  }
}


/*
  Backward of integrated_conv.  Each thread group handles a single channel (equal
  to blockIdx.x), and loops over patches of the output and over the image n
  within the batch (different thread groups may be responsible for different
  subsets of patches and/or images, see docs of gridDim below).

  If you want to understand this code, you should first understand the forward
  code.  Here are some points to understand how this works:

  First, understand the difference between the patch of size patchH by
  patchW, which is the basic patch size that is related to the blockDim.x,
  and the padded patch size ppatchH and ppatchW, where:
     ppatchH = patchH + kH - 1
     ppatchW = patchW + kW - 1.

  In the forward pass, we dealt with a patch of output and a padded patch of
  input.  In this backward-pass code, when computing the `grad_input` we deal
  with a patch of input and a padded patch of output (this ensures that
  different thread-blocks write to distinct patches of `grad_input`).  But this
  approach is not sufficient to update `grad_pos_add` and `grad_pos_mul`,
  because it's possible for elements of the zero-padding of `input` to
  contribute to `grad_pos_add` and `grad_pos_mul`.  So when computing the
  gradients for those quantities, we actually use a padded input patch and an
  un-padded output patch.  This requires that we load into shared memory the
  padded versions of both input and grad_output.


  Template args:
      scalar_t: the floating-point type, e.g. float, double, maybe half.

  Args:
      input [in]:  input image, shape (N, 2*C, H, W)
      pos_add [in]:  positional encoding, additive part,  shape (C, kH, kW)
      pos_mul [in]:  positional encoding, multiplicative part, shape (C, kH, kW)
      grad_output [in]: the gradient w.r.t. the output of the forward pass, shape (N, C, H, W)
      grad_input [out]: the gradient w.r.t. the input, of shape N, 2*C, H, W
      grad_pos_add [out]: the gradient w.r.t. pos_add, indexed [block][c][kh][kw],
            of shape num_blocks, C, kH, kW,
            where `block` is an index we'll later sum over, that corresponds to
            the identity of the thread-block (except, not including the channel
            dimension == gridDim.x).  So, block == blockIdx.z * gridDim.y + blockIdx.y,
            and num_blocks == gridDim.y * gridDim.z.
      grad_pos_mul [out]: the gradient w.r.t. pos_mul, like grad_pos_add above.
      patchH: the height of the patch size this kernel operates on (prior to padding)
      patchW: the width of the patch size this kernel operates on (prior to padding)
      threads_per_pixel: the number of threads assigned to compute each pixel
              of grad_input.  Require patchH * patchW * threads_per_pixel <= blockDim.x
              and threads_per_pixel must be a power of 2 in the interval [1,32].
      threads_per_kernel_pos: the number of threads assigned to compute each kernel
              position of grad_pos_add and grad_pos_mul.
              Require kH * kW * threads_per_kernel_pos <= blockDim.x,
              and threads_per_kernel_pos must be a power of 2 in the interval [1,32].
              This requires that kH * kW must not be greater than 1024.

  Note: kH and kW must both be odd so that it's clear how to zero-pad.

  The thread-block should have one dimension (x); see docs for threads_per_pixel
  and threads_per_kernel_pos for requirements on blockDim.x.  Also, blockDim.x
  must be an exact multiple of 64, so we can divide the threads by 2 and they
  will be in different warps.

  The requirements on the grid dimension are:
       gridDim.x == num-channels C (required)
       gridDim.y <= num-patches per image (recommended)
       gridDim.z <= batch-size N (recommended)
  When we invoke this kernel, we'll invoke it as:
   integrated_conv_forward<<<gridDim, blockDim, bytesShared, stream>>>
  where bytesShared is the number of bytes needed in `extern_buf`:

   bytesShared = sizeof(shared_t) * numel, where
    numel = 4 * (kH * kW) + 3 * (ppatchH * ppatchW) + blockDim.x
 */


template <typename scalar_t>
__global__
void integrated_conv_kernel_backward(
    torch::PackedTensorAccessor32<scalar_t, 4> input,  // N, 2*C, H, W
    torch::PackedTensorAccessor32<scalar_t, 3> pos_add,  // C, kH, kW
    torch::PackedTensorAccessor32<scalar_t, 3> pos_mul,  // C, kH, kW
    torch::PackedTensorAccessor32<scalar_t, 4> grad_output,  // N, C, H, W
    torch::PackedTensorAccessor32<scalar_t, 4> grad_input,  // N, 2*C, H, W
    torch::PackedTensorAccessor32<scalar_t, 4> grad_pos_add, // block, C, kH, kW, see above for `block`
    torch::PackedTensorAccessor32<scalar_t, 4> grad_pos_mul, // block, C, kH, kW, see above for `block`
    int patchH,  // non-padded patch height
    int patchW,  // non-padded patch width
    int threads_per_pixel,
    int threads_per_kernel_pos) {

  const int H = input.size(2),
      W = input.size(3),
      kH = pos_add.size(1),
      kW = pos_add.size(2),
      npatchH = (H + patchH - 1) / patchH,  // num patches in vertical dim
      npatchW = (W + patchW - 1) / patchW,  // num patches in horizontal dim
      npatch = npatchH * npatchW;  // total number of patches per image

  // Channel index.
  const int c = blockIdx.x;
  // We don't need to check the range of `c` because we set gridDim.x to the
  // exact number of channels.

  const int ppatchH = patchH + kH - 1,  // ppatchH is the padded patch height.
      ppatchW = patchW + kW - 1,  // ppatchW is the padded patch width
      patch_size = patchH * patchW,  // un-padded patch size
      ppatch_size = ppatchH * ppatchW;  // padded patch size

  // `extern_buf` is general-purpose shared memory, which we'll divide between
  // various buffers.

  // these are pointers to __shared__ memory; the compiler should
  // be able to figure this out.
  scalar_t
      *pos_add_buf = (scalar_t*)extern_buf,     // pos_add positional-encoding / kernel parameters,
                                // indexed [kh*kW + kw] where kh and kw are vertical
                                // and horizontal positions in the kernel.
      *pos_mul_buf = pos_add_buf + (kH * kW), // pos_mul positional-encoding / kernel parameters,
                                              // indexed [kh*kW + kw] where kh and kw are vertical
                                              // and horizontal positions in the kernel.
      *src_img_buf = pos_mul_buf + (kH * kW),    // version of input image that relates to source position,
                             // of size [ppatch_size], indexed [h*ppatchW + w],
                             // where the 'h' and 'w' indexes are into the zero-padded input
                             // image.
      *dest_img_buf = src_img_buf + ppatch_size,  // version of input image that relates to destinatioon position
      *grad_output_buf = src_img_buf + ppatch_size, // output gradient for padded patch, indexed [h*ppatchW + w]
      *grad_pos_add_buf = grad_output_buf + ppatch_size,  // total grad for pos_add for this thread block, indexed [kh*kW + kw]
      *grad_pos_mul_buf = grad_pos_add_buf + (kH * kW),  // total grad for pos_mul for this thread block, indexed [kh*kW + kw]
      *reduce_buf = grad_pos_mul_buf + (kH * kW);  // buffer for reduction over threads, size == blockDim.x


  // pos_in_patch will be interpreted as h_in_patch * patchW + w_in_patch.
  int pos_in_patch = threadIdx.x / threads_per_pixel;

  // Load parts of the kernel parameters pos_add and pos_mul into shared memory,
  // in pos_add_buf and pos_mul_buf; zero the corresponding gradient buffers.
  // We know that blockDim.x >= kH * kW, see threads_per_kernel_pos.
  if (threadIdx.x < kH * kW) {
    int i = threadIdx.x;
    int kh = i / kW, kw = i % kW;
    pos_add_buf[i] = pos_add[c][kh][kw];
    pos_mul_buf[i] = pos_mul[c][kh][kw];
    grad_pos_add_buf[i] = 0.0;
    grad_pos_mul_buf[i] = 0.0;
  }

  // n is the index within the batch of images.  Loop to make sure we cover all
  // images in the batch.  input.size(0) is the batch size N.  All threads in
  // the thread-block loop the same number of times.
  for (int n = blockIdx.z; n < input.size(0); n += gridDim.z) {

    // Loop over the patch within the output image.  All threads in the
    // thread-block loop the same number of times.
    for (int patch_idx = blockIdx.y; patch_idx < npatch; patch_idx += gridDim.y) {
      // (patch_h_offset, patch_w_offset) are the (vertical, horizontal) indexes
      // of the lowest-numbered pixel in the *un-padded* patch that this thread
      // block is responsible for.  (We'll actualy be loading the padded patches
      // into memory, so be careful).
      int patch_h_offset = (patch_idx / npatchW) * patchH,
          patch_w_offset = (patch_idx % npatchW) * patchW;

      // This __syncthreads() is only necessary if we have already looped at
      // least once over n or patch_idx: it's in case other threads are still
      // using the `src_img_buf` or `dst_img_buf` buffers for a previous patch.
      __syncthreads();

      // Load the 'src' and 'dest' versions of the padded patch into
      // shared-memory buffers, and also the output gradient.
      for (int i = threadIdx.x % (blockDim.x / 2); i < ppatch_size; i += (blockDim.x / 2)) {
        int h_in_ppatch = i / ppatchW,
            w_in_ppatch = i % ppatchW;
        int h = patch_h_offset + h_in_ppatch - (kH / 2),  // kH / 2 is offset due to padding
            w = patch_w_offset + w_in_ppatch - (kW / 2);

        if (threadIdx.x < blockDim.x / 2) {  // The first half of the threads of the block
                                             // load `input`
          scalar_t src_val = scalar_t(0),
              dest_val = scalar_t(0);
          if ((unsigned int)h < (unsigned int)H &&  // h >= 0 && h < H.
              (unsigned int)w < (unsigned int)W) {  // w >= 0 && w < W
            int C = grad_output.size(1);
            src_val = input[n][c][h][w];
            dest_val = input[n][c + C][h][w];
          }
          src_img_buf[i] = src_val;
          dest_img_buf[i] = dest_val;
        } else {  // second half of threads load `grad_output`.  We require
                  // blockDim.x be an even multiple of the warp size, so there
                  // is no warp divergence here.
          scalar_t grad_output_val = scalar_t(0);
          if ((unsigned int)h < (unsigned int)H &&
              (unsigned int)w < (unsigned int)W)
            grad_output_val = grad_output[n][c][h][w];
          grad_output_buf[i] = grad_output_val;
        }
      }
      // make sure all threads haave written to `src_img_buf`, `dest_img_buf` and
      // `grad_output_buf`.
      __syncthreads();

      scalar_t grad_input_src_sum = 0.0,  // grad for channel c, for our pixel
                                          // of `input` (contribution of this
                                          // thread)
          grad_input_dest_sum = 0.0;   // grad for channel c + C, for our pixel
                                       // of `input` (contribution of this thread)
      if (pos_in_patch < patch_size) {
        // This block computes `grad_input_sum`.
        // The num-threads for the backward kernel may not be an exact multiple
        // of patch_size, wo we need the if-guard.

        int h_in_patch = pos_in_patch / patchW,
            w_in_patch = pos_in_patch % patchW,
            h_in_ppatch = h_in_patch + kH / 2,
            w_in_ppatch = w_in_patch + kW / 2,
            pos_in_ppatch = h_in_ppatch * ppatchW + w_in_ppatch;

        // this_dest_val is the `destination` version of our current pixel; this
        // is an input.  It gets added to each src pixel, prior to the relu, in
        // the loop below.
        // this_src_val is the `src` version of our current pixel; it contributes
        // to the outputs of other pixels.
        scalar_t this_dest_val = dest_img_buf[pos_in_ppatch],
            this_src_val = src_img_buf[pos_in_ppatch];

        for (int pos_in_kernel = threadIdx.x % threads_per_pixel;
             pos_in_kernel < (kH * kW);
             pos_in_kernel += threads_per_pixel) {

          int h_in_kernel = pos_in_kernel / kW,
              w_in_kernel = pos_in_kernel % kW;

          // This is actually more like cross-correlation, as we don't have a
          // negative sign on the h and w indexes in the kernel.

          int src_h_in_ppatch = h_in_patch + h_in_kernel,
              src_w_in_ppatch = w_in_patch + w_in_kernel;
          int src_pos_in_ppatch = src_h_in_ppatch * ppatchW + src_w_in_ppatch;

          scalar_t src_val = src_img_buf[src_pos_in_ppatch],
              pos_add_val = pos_add_buf[pos_in_kernel],
              pos_mul_val = pos_mul_buf[pos_in_kernel];
          scalar_t relu = (src_val + this_dest_val + pos_add_val);
          if (relu >= 0.0) {
            scalar_t this_grad_output = grad_output_buf[pos_in_ppatch];
            grad_input_dest_sum += this_grad_output * pos_mul_val;
          }
          // To compute a contribution to "this_input_src_grad", we need to consider the
          // contribution to the destination pixel that it would have contributed to
          // with this same offset.
          int dest_h_in_ppatch = h_in_patch + (kH - 1) - h_in_kernel,
              dest_w_in_ppatch = w_in_patch + (kW - 1) - w_in_kernel,
              dest_pos_in_ppatch = dest_h_in_ppatch * ppatchW + dest_w_in_ppatch;
          scalar_t dest_val = dest_img_buf[dest_pos_in_ppatch];
          relu = dest_val + this_src_val + pos_add_val;
          if (relu >= 0.0) {
            scalar_t dest_grad_output = grad_output_buf[dest_pos_in_ppatch];
            grad_input_src_sum += dest_grad_output * pos_mul_val;
          }
        }
      }
      // Aggregate `grad_input_src_sum` over threads, if needed; and write the
      // result to `grad_input`.
      int h = patch_h_offset + pos_in_patch / patchW,
          w = patch_w_offset + pos_in_patch % patchW;

      if (h < H && w < W) {
        grad_input_src_sum = tiled_warp_reduce_sum(threads_per_pixel,
                                                   reduce_buf,
                                                   grad_input_src_sum);
        grad_input_dest_sum = tiled_warp_reduce_sum(threads_per_pixel,
                                                    reduce_buf,
                                                    grad_input_dest_sum);
        if (threadIdx.x % threads_per_pixel == 0) {
          grad_input[n][c][h][w] = grad_input_src_sum;
          int C = grad_output.size(1);
          grad_input[n][c + C][h][w] = grad_input_dest_sum;
        }
      }

      // OK, we are done computing grad_input for this patch.  Now
      // we need to contribute the contributions to grad_pos_add_buf
      // and grad_pos_mul_buf for this patch.
      // 0 <= pos_in_kernel < (kH * kW).
      int pos_in_kernel = threadIdx.x / threads_per_kernel_pos;
      scalar_t this_grad_pos_add = 0.0,
              this_grad_pos_mul = 0.0;
      if (pos_in_kernel < (kH * kW)) {
        int kh = pos_in_kernel / kW,
            kw = pos_in_kernel % kW;

        // This group of (threads_per_kernel_pos) threads is responsible
        // for position (kh, kw) in the kernel; we iterate over the patch.
        scalar_t pos_add_val = pos_add_buf[pos_in_kernel],
            pos_mul_val = = pos_mul_buf[pos_in_kernel];

        for (int pos_in_patch = threadIdx.x % threads_per_kernel_pos;
             pos_in_patch < patch_size; pos_in_patch += threads_per_kernel_pos) {
          // We are working out the contribution to the gradients for pos_add
          // and pos_mul; we let `pos_in_patch` correspond to the *output*
          // position, and work out the input position based on gthe kernel position.

          int h_in_patch = pos_in_patch / patchH,
              w_in_patch = pos_in_patch / patchW;

          // pos_in_ppatch is the position in the padded patch corresponding to
          // `pos_in_patch`.
          int pos_in_ppatch = (h_in_patch + kH / 2) * ppatchW + (w_in_patch + kW / 2);
          scalar_t dest_val = dest_img_buf[pos_in_ppatch];
          int offset_pos_in_ppatch = (h_in_patch + kh) * ppatchW + (w_in_patch + kw);
          scalar_t src_val = src_img_buf[offset_pos_in_ppatch];

          scalar_t relu = dest_val + src_val + pos_add_val;
          if (relu >= 0.0) {
            scalar_t this_grad_output = grad_output_buf[pos_in_ppatch];
            this_grad_pos_add += this_grad_output * pos_mul_val;
            this_grad_pos_mul += this_grad_output * relu;
          }
        }
        this_grad_pos_add = tiled_warp_reduce_sum(
            threads_per_kernel_pos, reduce_buf, this_grad_pos_add);
        this_grad_pos_mul = tiled_warp_reduce_sum(
            threads_per_kernel_pos, reduce_buf, this_grad_pos_mul);
        if (threadIdx.x % threads_per_kernel_pos == 0) {
          grad_pos_add_buf[pos_in_kernel] = this_grad_pos_add;
          grad_pos_mul_buf[pos_in_kernel] = this_grad_pos_mul;
        }
      }
    }
  }

  int block = blockIdx.z * gridDim.y + blockIdx.y;

  int kernel_pos = threadIdx.x;
  if (kernel_pos < (kH * kW)) {
    int kh = kernel_pos / kW,
        kw = kernel_pos % kW;
    grad_pos_add[block][c][kh][kw] = grad_pos_add_buf[kernel_pos];
    grad_pos_mul[block][c][kh][kw] = grad_pos_mul_buf[kernel_pos];
  }
}








torch::Tensor integrated_conv_cuda(torch::Tensor input,
                                   torch::Tensor pos_add,
                                   torch::Tensor pos_mul) {
  TORCH_CHECK(input.dim() == 4, "input must be 4-dimensional");
  TORCH_CHECK(pos_add.dim() == 3, "pos_add must be 3-dimensional.");
  TORCH_CHECK(pos_mul.dim() == 3, "pos_add must be 3-dimensional.");
  TORCH_CHECK(input.device().is_cuda(), "Input must be a CUDA tensor");
  const int N = input.size(0),
      C = input.size(1) / 2,
      H = input.size(2),
      W = input.size(3),
      kH = pos_add.size(1),
      kW = pos_add.size(2);
  TORCH_CHECK(kH % 2 == 1 && kW % 2 == 1);
  TORCH_CHECK(input.size(1) % 2 == 0, "Input must have even num-channels");
  TORCH_CHECK(pos_add.size(0) == C && pos_mul.size(0) == C &&
              pos_mul.size(1) == kH && pos_mul.size(2) == kW,
              "Input sizes mismatch.");
  TORCH_CHECK(pos_add.device() == input.device() &&
              pos_mul.device() == pos_add.device(),
              "Input devices mismatch");
  auto scalar_t = input.scalar_type();
  TORCH_CHECK(pos_add.scalar_type() == scalar_t &&
              pos_mul.scalar_type() == scalar_t,
              "Input dtypes mismatch");

  torch::Tensor output = torch::empty({N, C, H, W},
                                      torch::TensorOptions().dtype(scalar_t).device(input.device()));


  // Work out the configuration to call the kernel with..
  int patchH = std::min(H, kH),  // output patch height
      patchW = std::min(W, kW);  // output patch width
  // We don't want the height or width of the patch to be less than the kernel
  // width, or the padding will make the input-patch size more than twice the
  // output-patch size.
  // We aim for the output-patch size to be more than 128; this is not something
  // very exact, but it roughly corresponds to us wanting to have up to 4 threads
  // per output pixel, and the limitation of 512 threads per thread-block which
  // we impose so that we can run on architectures with little shared memory.
  while (patchW < W && patchH * (patchW + 1) <= 128)
    patchW++;
  while(patchH < H && (patchH + 1) * patchW <= 128)
    patchH++;

  // We are assuming that the thread-block size can be as large as 512; this
  // works even on old CUDA architectures.
  int threads_per_opixel;
  if (patchH * patchW * 4 <= 512 && (kH * kW) > 16)
    threads_per_opixel = 4;
  else if (patchH * patchW * 2 <= 512 && (kH * kW) > 8)
    threads_per_opixel = 2;
  else
    threads_per_opixel = 1;

  int input_patchH = patchH + kH - 1,
         input_patchW = patchW + kW - 1,
         input_patch_size = input_patchH * input_patchW;

  int threads_per_block = patchH * patchW * threads_per_opixel;

  int buffer_numel = 2 * (kH * kW) + std::max<int>(threads_per_block,
                                                   input_patch_size);

  int num_patches_H = (H + patchH - 1) / patchH,
      num_patches_W = (W + patchW - 1) / patchW,
      num_patches = num_patches_H * num_patches_W;

  // gridDim.x == C.
  int num_blocks_patch = 1,  // gridDim.y.
       num_blocks_batch = 1;  // gridDim.z
  while (C * num_blocks_patch <= 256 &&
         num_blocks_patch * 2 <= num_patches)
    num_blocks_patch *= 2;
  if (C * num_patches <= 512)
    num_blocks_patch = num_patches;
  while (C * num_blocks_patch * num_blocks_batch <= 512 &&
         num_blocks_batch * 2 <= N)
    num_blocks_batch *= 2;
  if (C * num_blocks_patch * N <= 1024)
    num_blocks_batch = N;

  assert(num_blocks_patch <= num_patches && num_blocks_batch <= N);

  std::cout << "N,C,H,W=" << N << "," << C << "," << H << "," << W
            << "; kW,kH=" << kW << "," << kH
            << "; patchH,patchW=" << patchH << ","
            << patchW << ", num_blocks_patch="
            << num_blocks_patch << ", num_blocks_batch="
            << num_blocks_batch
            << ", threads_per_opixel=" << threads_per_opixel
            << ", threads_per_block=" << threads_per_block
            << std::endl;

  dim3 gridDim(C, num_blocks_patch, num_blocks_batch);
  // blockDim is scalar, just threads_per_block.
  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "integrated_conv_kernel", ([&] {
        integrated_conv_kernel<scalar_t><<<gridDim, threads_per_block, sizeof(scalar_t) * buffer_numel, at::cuda::getCurrentCUDAStream()>>>(
              input.packed_accessor32<scalar_t, 4>(),
              pos_add.packed_accessor32<scalar_t, 3>(),
              pos_mul.packed_accessor32<scalar_t, 3>(),
              output.packed_accessor32<scalar_t, 4>(),
              patchH,
              patchW);
      }));
  return output;
}



std::vector<torch::Tensor> integrated_conv_backward_cuda(torch::Tensor input,
                                                         torch::Tensor pos_add,
                                                         torch::Tensor pos_mul,
                                                         torch::Tensor grad_output) {
  TORCH_CHECK(input.dim() == 4, "input must be 4-dimensional");
  TORCH_CHECK(pos_add.dim() == 3, "pos_add must be 3-dimensional.");
  TORCH_CHECK(pos_mul.dim() == 3, "pos_add must be 3-dimensional.");
  TORCH_CHECK(input.device().is_cuda(), "Input must be a CUDA tensor");
  const int N = input.size(0),
      C = input.size(1) / 2,
      H = input.size(2),
      W = input.size(3),
      kH = pos_add.size(1),
      kW = pos_add.size(2);
  TORCH_CHECK(kH % 2 == 1 && kW % 2 == 1);
  TORCH_CHECK(input.size(1) % 2 == 0, "Input must have even num-channels");
  TORCH_CHECK(pos_add.size(0) == C && pos_mul.size(0) == C &&
              pos_mul.size(1) == kH && pos_mul.size(2) == kW,
              "Input sizes mismatch.");
  TORCH_CHECK(pos_add.device() == input.device() &&
              pos_mul.device() == pos_add.device(),
              "Input devices mismatch");
  auto scalar_t = input.scalar_type();
  TORCH_CHECK(pos_add.scalar_type() == scalar_t &&
              pos_mul.scalar_type() == scalar_t,
              "Input dtypes mismatch");
  TORCH_CHECK(grad_output.dim() == 4 && grad_output.size(0) == N
              && grad_output.size(1) == C && grad_output.size(2) == H
              && grad_output.size(3) == W);


  // Work out the configuration to call the kernel with..
  int patchH = std::min(H, kH),  // output patch height
      patchW = std::min(W, kW);  // output patch width
  // We don't want the height or width of the patch to be less than the kernel
  // width, or the padding will make the input-patch size more than twice the
  // output-patch size.
  // We aim for the output-patch size to be more than 128; this is not something
  // very exact, but it roughly corresponds to us wanting to have up to 4 threads
  // per output pixel, and the limitation of 512 threads per thread-block which
  // we impose so that we can run on architectures with little shared memory.
  while (patchW < W && patchH * (patchW + 1) <= 128)
    patchW++;
  while(patchH < H && (patchH + 1) * patchW <= 128)
    patchH++;

  // We are assuming that the thread-block size can be as large as 512; this
  // works even on old CUDA architectures.
  int threads_per_pixel;
  if (patchH * patchW * 4 <= 512 && (kH * kW) > 8)
    threads_per_pixel = 4;
  else if (patchH * patchW * 2 <= 512 && (kH * kW) > 4)
    threads_per_pixel = 2;
  else
    threads_per_pixel = 1;

  int threads_per_block = patchH * patchW * threads_per_pixel;
  // round threads_per_block up to a multiple of 64.  We need it to be
  // equivalent to an even number of warps, because at one point we divide the
  // threads into two halves and we want them to be an even number of warps.
  threads_per_block = 64 * ((threads_per_block + 63) / 64);

  {
    // If it's possible to increase the patch width or height while not exceeding
    // this number of threads, do so.  (This is a small optimization).
    int patchW_old = patchW;
    while (patchH * (patchW + 1) * threads_per_pixel <= threads_per_block)
      patchW++;
    // If the above change to patchW did not actually reduce the number of patches
    // needed to cover the image, gthen there is no point to the change; and it
    // increases the shared-memory requirement, so revert it.
    if ((W + patchW_old - 1) / patchW_old == (W + patchW - 1) / patchW)
      patchW = patchW_old;
    int patchH_old = patchH;
    while ((patchH + 1) * patchW * threads_per_pixel <= threads_per_block)
      patchH++;
    if ((H + patchH_old - 1) / patchH_old == (H + patchH - 1) / patchH)
      patchH = patchH_old;
  }


  int threads_per_kernel_pos = 1;
  while (threads_per_kernel_pos < 32 &&
         threads_per_kernel_pos * 2 * kH * kW <= threads_per_block)
    threads_per_kernel_pos *= 2;

  // dimensions of padded patches
  int ppatchH = patchH + kH - 1,
       ppatchW = patchW + kW - 1,
   ppatch_size = ppatchH * ppatchW;

  int buffer_numel = 4 * (kH * kW) + 3 * ppatch_size + threads_per_block;

  int num_patches_H = (H + patchH - 1) / patchH,
      num_patches_W = (W + patchW - 1) / patchW,
      num_patches = num_patches_H * num_patches_W;

  // gridDim.x == C.
  int num_blocks_patch = 1,  // gridDim.y.  should not be more
      num_blocks_batch = 1;  // gridDim.z
  // We have a rough target of no more than 256 thread-groups.
  while (C * num_blocks_patch * 2 <= 256 &&
         num_blocks_patch * 2 <= num_patches)
    num_blocks_patch *= 2;
  if (C * num_patches <= 512)
    num_blocks_patch = num_patches;
  while (C * num_blocks_patch * num_blocks_batch * 2 <= 256 &&
         num_blocks_batch * 2 <= N)
    num_blocks_batch *= 2;

  assert(num_blocks_patch <= num_patches && num_blocks_batch <= N);
  assert(patchH * patchW * threads_per_pixel <= threads_per_block);
  assert(kH * kW * threads_per_kernel_pos <= threads_per_block);

  std::cout << "[backward:] N,C,H,W=" << N << "," << C << "," << H << "," << W
            << "; kW,kH=" << kW << "," << kH
            << "; patchH,patchW=" << patchH << ","
            << patchW << ", num_blocks_patch="
            << num_blocks_patch << ", num_blocks_batch="
            << num_blocks_batch
            << ", threads_per_pixel=" << threads_per_pixel
            << ", threads_per_kernel_pos=" << threads_per_kernel_pos
            << ", threads_per_block=" << threads_per_block
            << ", buffer_numel=" << buffer_numel
            << std::endl;

  int num_blocks = num_blocks_patch * num_blocks_batch;

  torch::Tensor grad_input = torch::zeros({N, 2*C, H, W},
                                          torch::TensorOptions().dtype(scalar_t).device(input.device())),
      grad_pos_add = torch::zeros({num_blocks, C, kH, kW},
                                  torch::TensorOptions().dtype(scalar_t).device(input.device())),
      grad_pos_mul = torch::zeros({num_blocks, C, kH, kW},
                                  torch::TensorOptions().dtype(scalar_t).device(input.device()));


  dim3 gridDim(C, num_blocks_patch, num_blocks_batch);
  // blockDim is scalar, just threads_per_block.
  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "integrated_conv_kernel", ([&] {
        integrated_conv_kernel<scalar_t><<<gridDim, threads_per_block, sizeof(scalar_t) * buffer_numel, at::cuda::getCurrentCUDAStream()>>>(
              input.packed_accessor32<scalar_t, 4>(),
              pos_add.packed_accessor32<scalar_t, 3>(),
              pos_mul.packed_accessor32<scalar_t, 3>(),
              grad_output.packed_accessor32<scalar_t, 4>(),
              grad_input.packed_accessor32<scalar_t, 4>(),
              grad_pos_add.packed_accessor32<scalar_t, 4>(),
              grad_pos_mul.packed_accessor32<scalar_t, 4>(),
              patchH,
              patchW,
              threads_per_pixel,
              threads_per_kernel_pos);
      }));
  grad_pos_add = at::sum(grad_pos_add, {0});
  grad_pos_mul = at::sum(grad_pos_mul, {0});

  return std::vector<torch::Tensor>({grad_input, grad_pos_add, grad_pos_mul});
}
