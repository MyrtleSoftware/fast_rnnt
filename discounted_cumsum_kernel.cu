#include "hip/hip_runtime.h"
#include <torch/extension.h>


template <typename scalar_t>
__device__ __forceinline__ scalar_t discounted_sum_pow(scalar_t a, scalar_t b, scalar_t gamma, int power) {
    return a + b * pow(gamma, scalar_t(power));
}


__inline__
int log2ceil(int x) {
    return (int)ceil(log2((float)x));
}


template <typename scalar_t>
__global__ void discounted_cumsum_right_kernel_minthreads_stage(
        torch::PackedTensorAccessor32<scalar_t, 2> x,
        const scalar_t gamma,
        int stage
) {
    // Pros: Minimum required number of threads, assigns them dynamically to respective positions upon each iteration.
    // Cons: Uncoalesced writes.

    const int len = x.size(1);
    const int threadidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (threadidy >= x.size(0)) {
        return;
    }

    int gr_prev_stride = 1 << stage;
    int gr_cur_stride = gr_prev_stride << 1;

    int gr_of_thread = threadidx >> stage;
    int thread_in_gr = threadidx - (gr_of_thread << stage);

    int change_pos = gr_of_thread * gr_cur_stride + thread_in_gr;
    int discounted_pos = gr_of_thread * gr_cur_stride + gr_prev_stride;
    int discount_power = gr_prev_stride - thread_in_gr;

    if (change_pos >= len || discounted_pos >= len) {
        return;
    }

    x[threadidy][change_pos] = discounted_sum_pow(
        x[threadidy][change_pos],
        x[threadidy][discounted_pos],
        gamma,
        discount_power
    );
}


template <typename scalar_t>
__global__ void discounted_cumsum_right_kernel_coalesced_stage(
        torch::PackedTensorAccessor32<scalar_t, 2> x,
        const scalar_t gamma,
        int stage
) {
    // Pros: Coalesced writes.
    // Cons: Threads allocated statically per each element. Half of threads idles upon each iteration.

    const int len = x.size(1);
    const int threadidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (threadidx >= len || threadidy >= x.size(0)) {
        return;
    }

    int gr_prev_stride = 1 << stage;
    int gr_cur_stride = gr_prev_stride << 1;

    int gr_of_thread = threadidx >> (stage + 1);
    int thread_in_gr = threadidx - (gr_of_thread << (stage + 1));

    int change_pos = threadidx;
    int discounted_pos = gr_of_thread * gr_cur_stride + gr_prev_stride;
    int discount_power = gr_prev_stride - thread_in_gr;

    if (thread_in_gr >= gr_prev_stride || discounted_pos >= len) {
        return;
    }

    x[threadidy][change_pos] = discounted_sum_pow(
        x[threadidy][change_pos],
        x[threadidy][discounted_pos],
        gamma,
        discount_power
    );
}


torch::Tensor discounted_cumsum_right_minthreads(torch::Tensor x, double gamma) {
    // Pros: Minimum required number of threads, assigns them dynamically to respective positions upon each iteration.
    // Cons: Uncoalesced writes.

    TORCH_CHECK(x.type().is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");
    TORCH_CHECK(x.dim() == 2, "Input must be 2-dimensional");
    TORCH_CHECK(0.0 <= gamma && gamma <= 1.0, "Gamma must be in the range [0,1]");

    if (x.size(1) == 0) {
        return x;
    }

    auto y = x.clone();

    const int threads = 32;
    const int nstages = log2ceil(x.size(1));
    const int threads_total_x = 1 << (nstages - 1);
    const dim3 blocks((threads_total_x + threads - 1) / threads, x.size(0));

    for (int stage=0; stage<nstages; stage++) {
        AT_DISPATCH_FLOATING_TYPES(x.type(), "discounted_cumsum_right_kernel_minthreads_stage", ([&] {
            discounted_cumsum_right_kernel_minthreads_stage<scalar_t><<<blocks, threads>>>(
                y.packed_accessor32<scalar_t, 2>(),
                scalar_t(gamma),
                stage
            );
        }));
    }

    return y;
}


torch::Tensor discounted_cumsum_right_coalesced(torch::Tensor x, double gamma) {
    // Pros: Coalesced writes.
    // Cons: Threads allocated statically per each element. Half of threads idles upon each iteration.

    TORCH_CHECK(x.type().is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");
    TORCH_CHECK(x.dim() == 2, "Input must be 2-dimensional");
    TORCH_CHECK(0.0 <= gamma && gamma <= 1.0, "Gamma must be in the range [0,1]");

    if (x.size(1) == 0) {
        return x;
    }

    auto y = x.clone();

    const int threads = 32;
    const int nstages = log2ceil(x.size(1));
    const dim3 blocks((x.size(1) + threads - 1) / threads, x.size(0));

    for (int stage=0; stage<nstages; stage++) {
        AT_DISPATCH_FLOATING_TYPES(x.type(), "discounted_cumsum_right_kernel_coalesced_stage", ([&] {
            discounted_cumsum_right_kernel_coalesced_stage<scalar_t><<<blocks, threads>>>(
                y.packed_accessor32<scalar_t, 2>(),
                scalar_t(gamma),
                stage
            );
        }));
    }

    return y;
}
