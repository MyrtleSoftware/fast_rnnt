#include "hip/hip_runtime.h"
#include <torch/extension.h>


template <typename scalar_t>
__device__ __forceinline__
scalar_t discounted_sum_pow(scalar_t a, scalar_t b, scalar_t gamma, int power) {
    return a + b * pow(gamma, scalar_t(power));
}


enum SumDirection {
    SUM_RIGHT,
    SUM_LEFT
};


template <SumDirection d>
__device__ __forceinline__
void resolve_positions(
        const int &gr_prev_stride, const int &gr_cur_stride, const int &gr_of_thread, const int &thread_in_gr,
        int &change_pos, int &discounted_pos, int &discount_power
);


template <>
__device__ __forceinline__
void resolve_positions<SUM_RIGHT>(
        const int &gr_prev_stride, const int &gr_cur_stride, const int &gr_of_thread, const int &thread_in_gr,
        int &change_pos, int &discounted_pos, int &discount_power
) {
    change_pos = gr_of_thread * gr_cur_stride + thread_in_gr;
    discounted_pos = gr_of_thread * gr_cur_stride + gr_prev_stride;
    discount_power = gr_prev_stride - thread_in_gr;
}


template <typename scalar_t, SumDirection d>
__global__
void discounted_cumsum_kernel_stage(
        torch::PackedTensorAccessor32<scalar_t, 2> x,
        const scalar_t gamma,
        int stage
) {
    const int len = x.size(1);
    const int threadidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (threadidy >= x.size(0)) {
        return;
    }

    int gr_prev_stride = 1 << stage;
    int gr_cur_stride = gr_prev_stride << 1;

    int gr_of_thread = threadidx >> stage;
    int thread_in_gr = threadidx - (gr_of_thread << stage);

    //int change_pos = gr_of_thread * gr_cur_stride + thread_in_gr;
    //int discounted_pos = gr_of_thread * gr_cur_stride + gr_prev_stride;
    //int discount_power = gr_prev_stride - thread_in_gr;

    int change_pos, discounted_pos, discount_power;
    resolve_positions<d>(
        gr_prev_stride, gr_cur_stride, gr_of_thread, thread_in_gr,
        change_pos, discounted_pos, discount_power
    );

    if (change_pos >= len || discounted_pos >= len) {
        return;
    }

    x[threadidy][change_pos] = discounted_sum_pow(
        x[threadidy][change_pos],
        x[threadidy][discounted_pos],
        gamma,
        discount_power
    );
}


inline
int log2ceil(int x) {
    return (int)ceil(log2((float)x));
}


template <SumDirection d>
torch::Tensor discounted_cumsum(torch::Tensor x, double gamma) {
    // Minimum required number of threads, assigns them dynamically to respective positions upon each iteration.
    // Results in uncoalesced writes, which is still faster than coalesced writes with half threads idling.

    TORCH_CHECK(x.type().is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(x.is_contiguous(), "Input must be contiguous");
    TORCH_CHECK(x.dim() == 2, "Input must be 2-dimensional");
    TORCH_CHECK(0.0 <= gamma && gamma <= 1.0, "Gamma must be in the range [0,1]");

    if (x.size(1) == 0) {
        return x;
    }

    auto y = x.clone();

    const int threads = 64;
    const int nstages = log2ceil(x.size(1));
    const int threads_total_x = 1 << (nstages - 1);
    const dim3 blocks((threads_total_x + threads - 1) / threads, x.size(0));

    for (int stage=0; stage<nstages; stage++) {
        AT_DISPATCH_FLOATING_TYPES(x.type(), "discounted_cumsum_kernel_stage", ([&] {
            discounted_cumsum_kernel_stage<scalar_t, d><<<blocks, threads>>>(
                y.packed_accessor32<scalar_t, 2>(),
                scalar_t(gamma),
                stage
            );
        }));
    }

    return y;
}


torch::Tensor discounted_cumsum_right(torch::Tensor x, double gamma) {
    return discounted_cumsum<SUM_RIGHT>(x, gamma);
}


//torch::Tensor discounted_cumsum_left(torch::Tensor x, double gamma) {
//}
